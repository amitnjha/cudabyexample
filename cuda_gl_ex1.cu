#include "hip/hip_runtime.h"
#define GL_GLEXT_PROTOTYPES

#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"

//  #include "cpu_anim.h"
#include ""

#include <GL/gl.h>
#include <GL/glext.h>
#include <GL/glut.h>
#include "cuda_gl_interop.h"
#include "cpu_bitmap.h"

// #include <gl.h>


#define GL_GLEXT_PROTOTYPES

#define DIM 512

GLuint bufferObj;
hipGraphicsResource *resource;


static void draw_func(void) {

  glDrawPixels(DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0);

  glutSwapBuffers();

}

static void key_func(unsigned char key, int x, int y){

  switch(key) {

  case 27:
    hipGraphicsUnregisterResource(resource);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
    glDeleteBuffers(1, &bufferObj);
    exit(0);
    
  }
}

__global__ void kernel(uchar4 *ptr){

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  float fx = x / (float)DIM - 0.5f;
  float fy = y / (float)DIM - 0.5f;

  unsigned char green = 128 + 127 * cos(abs(fx*100) - abs(fy*100));

  ptr[offset].x = 0;
  ptr[offset].y = green;
  ptr[offset].z = 0;
  ptr[offset].w = 255;
    
}


int main(int argc, char **argv){

  hipDeviceProp_t prop;
  int dev;

  memset( &prop, 0, sizeof(hipDeviceProp_t) );

  prop.major = 1;
  prop.minor = 0;

  hipChooseDevice(&dev, &prop);

  // cudaGLSetGLDevice(dev);

  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
  glutInitWindowSize(DIM, DIM);
  glutCreateWindow("bitmap");

  glGenBuffers(1, &bufferObj);
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
  glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, DIM * DIM * 4, NULL, GL_DYNAMIC_DRAW_ARB);

  hipGraphicsGLRegisterBuffer(&resource, bufferObj, cudaGraphicsMapFlagsNone);

  uchar4* devPtr;

  size_t size;

  hipGraphicsMapResources(1, &resource, NULL);
  hipGraphicsResourceGetMappedPointer( (void**)&devPtr, &size, resource);

  dim3 grids(DIM/16, DIM/16);
  dim3 threads(16,16);

  kernel<<<grids,threads>>>(devPtr);

  hipGraphicsUnmapResources(1, &resource, NULL);

  glutKeyboardFunc(key_func);
  glutDisplayFunc(draw_func);

  glutMainLoop();
  

}
