#include <stdio.h>
#include "hip/hip_runtime.h"





__global__
void HelloFromGPU()
{
    printf("********************************\n");
    printf("*        JETSON NANO           *\n");
    printf("*        PROPERTIES            *\n");
    printf("********************************\n");
}

int main ()
{
    HelloFromGPU<<<1,1>>>();
    hipDeviceSynchronize();
    
    int deviceId;
    hipGetDevice(&deviceId);
    
    hipDeviceProp_t properties;     
    hipGetDeviceProperties(&properties, deviceId);

    printf("Name:  %s \n", properties.name);
    printf("Multi Processor Count:  %d \n", properties.multiProcessorCount);
    printf("Max threads per block %d \n", properties.maxThreadsPerBlock);
    printf("Num of Blocks in x:  %d \n", properties.maxGridSize[0]);    
    printf("Num of Blocks in y:  %d \n", properties.maxGridSize[1]);
    printf("Num of Blocks in z:  %d \n", properties.maxGridSize[2]);
    printf("Compute Capability:  %d .%d \n", properties.major, properties.minor);
    printf("Name: %s\n", properties.name);
    printf("totalGlobalMem: %lu\n", properties.totalGlobalMem);
    printf("Integrated: %d\n", properties.integrated);
    return 0;
}



/*
int main(void){

  cudaDeviceProp prop;
  int count;

  printf("%d",cudaGetDeviceCount(&count));

  for( int i = 0; i < count; i++){

    cudaGetDeviceProperties(&prop,i);
    // do something with prop;
    
  }

}

*/
