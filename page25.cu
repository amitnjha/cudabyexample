#include <stdio.h>
#include <limits.h>
#include "hip/hip_runtime.h"



__global__ void add(int a, int b, int *c){

  *c = a + b;

}


int main(void){


  int c;
  int *dev_c;

  // HANDLE_ERROR(cudaMalloc( (void**)&dev_c, sizeof(int)) );
  hipMalloc( (void**)&dev_c, sizeof(int));

  add<<<1,1>>>(2,7, dev_c);

  /* HANDLE_ERROR(cudaMemcpy( &c,
			   dev_c,
			   sizeof(int),
			   cudaMemcpyDeviceToHost)); */

  hipMemcpy( &c,dev_c,  sizeof(int), hipMemcpyDeviceToHost);
  printf("2 + 7 = %d\n",c);

  hipFree(dev_c);

  return 0;

}
