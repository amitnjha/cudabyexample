#include <stdio.h>
#include "hip/hip_runtime.h"


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( 1 );}}
#define SIZE (100 * 1024 * 1024)

void* big_random_block( int size ) {
    unsigned char *data = (unsigned char*)malloc( size );
    HANDLE_NULL( data );
    for (int i=0; i<size; i++)
        data[i] = rand();

    return data;
}

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo){

  int i = threadIdx.x+ blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  while(i < size){

    atomicAdd( &(histo[buffer[i]]), 1);
    //histo[buffer[i]]+=1;
    i += stride;
    
  }

}


int main(void){

  unsigned char *buffer =  (unsigned char*)big_random_block(SIZE);
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  unsigned char *dev_buffer;
  unsigned int *dev_histo;

  hipMalloc((void**)&dev_buffer, SIZE);
  hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);

  hipMalloc((void**)&dev_histo, 256 * sizeof(int));
  hipMemset(dev_histo, 0, 256 * sizeof(int));
  
  // computation

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int blocks = prop.multiProcessorCount;
  histo_kernel<<<blocks*2, 256>>>(dev_buffer, SIZE, dev_histo);
  
  unsigned int histo[256];


  //copy back

  hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);

  hipEventRecord(stop,0);

  hipEventSynchronize(stop);

  float elapsedTime = 0;
  hipEventElapsedTime(&elapsedTime, start, stop);

  printf("Time to generate: %3.1f ms\n", elapsedTime);

  long histoCount  = 0;

  for (int i = 0; i < 256; i++){
    histoCount += histo[i];
  }

  printf("Histogram Sum :%ld\n", histoCount);

  for (int i = 0 ; i< SIZE; i++){
    histo[buffer[i]]--;
  }

  for(int i = 0; i<256; i++){

    if(histo[i] != 0)
      printf("Failure at %d\n", i);
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(dev_histo);
  hipFree(dev_buffer);
  free(buffer);
  return 0;
  
}



